#include "hip/hip_runtime.h"
#include "glnetwork.h"  //must be first
#include <math.h>
#include <iostream>
#include <vector>
#include <string>
#include <sstream>
#include <GL/glew.h>
#include <GL/glut.h>
#include <GL/freeglut.h>
#include <stim/math/vector.h>
#include <stim/gl/gl_spider.h>
#include <stim/gl/gl_texture.h>
#include <stim/gl/error.h>
#include <stim/visualization/camera.h>
#include <stim/visualization/glObj.h>

#include <stim/parser/arguments.h>
#ifdef TIMING
	#include <cstdio>
	#include <ctime>
#endif
#define _USE_MATH_DEFINES

//Vis and Coord Variables
#include<stim/math/constants.h>
	const float degtorad		= 360.0/(stim::TAU);
	stim::vec3<float> v_size(1.0, 1.0, 5.0);
	stim::vec3<float> i_size(1024.0, 1024.0, 98.0);
	stim::vec3<float> size(v_size[0]*i_size[0], v_size[1]*i_size[1], v_size[2]*i_size[2]);
	stim::vec<int> w_size(900,900);
	float GL[3] = {4000.0, 4000.0, 20000.0};
	stim::vec3<float> p;
	stim::vec3<float> up; 
	stim::vec3<float> d;  
	bool gui 			= false;
	int cyl = 0;
	


//Glut Variables
	float a = size[0]; 		//size of the volume.
        float b = size[1];		//y
        float c = size[2];		//z
        float z = 0.0;
	float curPos = 0.0;
	stim::camera cam;
	static bool button1		= false;
	static float mousePos[2]	= {0,0};
	static float prevmousePos[2]	= {0,0};
	
	stim::camera camSel;
        GLint lines[24] 	= {0,1,1,2,2,3,0,3,4,5,5,7,7,6,6,4,3,6,2,7,1,5,0,4};
	GLfloat vertexDrw[8][3];
	

//Runtime Variables
	
	GLuint texID;
        bool started = false;
	stim::gl_spider<float> spidey(200,100,50);
	std::vector<stim::vec3<float> > sl;
	std::vector<stim::vec3<float> > current;
        stim::vec3<float> temp(0.0, 0.0, 1.0);
	stim::vec3<float> ps;
	stim::vec3<float> ups; 
	stim::vec3<float> ds;  
	std::string sfname = "skel_new.txt";

//Testing Variables
#ifdef TIMING
	std::clock_t start;
	double duration;
#endif

	stim::glObj<float> skeleton; // glObject.
	stim::glnetwork<float> net;


//--------------------HELPER METHODS--------------------------------//
//------------------------------------------------------------------//
//Given a file name, load all the seedpoints.
//Depreciated
void
loadSeedPoints(std::string file)
{
	 
         std::ifstream myfile(file.c_str());
         string line;
         if(myfile.is_open())
         {
                 while (getline(myfile, line))
                 {
                         float x, y, z;
                         myfile >> x >> y >> z;
             		 sl.push_back(stim::vec<float>(
				((float) x)*0.6, 
				((float) y)*0.6, 
				((float) z)*1.0));
                 }
                 myfile.close();
        } else {
			std::cerr<<"failed" << std::endl;
	}
}
///updateVertices
///float a -- the new x coordinate for the basis of the cube.
///float b -- the new y coordinate for the basis of the cube.
///float c -- the new z coordinate for the basis of the cube.
void
updateCubeVertices(float aa, float bb, float cc)
{
	GLfloat temp[8][3] = {{bb, bb, z}, {z, bb, z}, {z, z, z},
                             {aa, z, z}, {aa, bb, cc}, {z, bb, cc},
                             {aa, z, cc}, {z, z, cc}};
	memcpy(&vertexDrw, &temp, sizeof(temp));
}


//Init 
//GLsizei xs -- (size in the x direction
//GLsizei xy -- (size in the y direction)
//Initializes all matrix and camera objects for GL, as well as loads the
//Seed points.

void 
glInit(GLsizei xs, GLsizei ys, int x = 512, int y = 512, int z = 298,
		 float dx = 0.6, float dy = 0.7, float dz = 1.0)
{

	 //Set all of thei global visuzalization variables.
	 ::v_size = stim::vec3<float>(dx, dy, dz);
	 ::i_size = stim::vec3<float>(x, y, z); 		
 	 ::size = stim::vec3<float>(v_size[0]*i_size[0], v_size[1]*i_size[1], v_size[2]*i_size[2]);

	 ::a = size[0]; 		//size of the volume.
         ::b = size[1];		//y
         ::c = size[2];		//z
	 updateCubeVertices(::a, ::b, ::c);
	 if(x >= 1024 || y >= 1024 || z >= 1024)
		 {GL[0] = 8000; GL[1] = 8000; GL[2] = 40000;}
	 

	 //Enable all the necessary GL options
	 //And clear all matrices for visualization.
         glEnable(GL_TEXTURE_3D);
         glEnable(GL_DEPTH_TEST);
         glTexEnvi(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);
	 glClearColor(0.0, 0.0, 0.0, 0.0);
	 glColor4d(0.0, 0.0, 0.0, 1.0);
   	 glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
         glShadeModel(GL_FLAT);
         glViewport(0, 0, xs, ys);
         glMatrixMode(GL_PROJECTION);
         glLoadIdentity();
	 glOrtho(0.0, GL[0], 0.0, GL[1], 0.0, GL[2]);
	 glMatrixMode(GL_TEXTURE);
	 glLoadIdentity();
         glMatrixMode(GL_MODELVIEW);
         glLoadIdentity();

	 //Attach the spider, to the texture and set all the properties of the searched volume.
         spidey.attachSpider(texID);
	 spidey.setSize(x, y, z);
	 spidey.setDims(dx,dy, dz);
         spidey.initCuda();
	 cam.setPosition(0.0, 0.0, 0.0);
	 cam.setFocalDistance(4000.0);
}


//---------------------------DRAW METHODS---------------------------//
//------------------------------------------------------------------//


//Draws a cube that surrounds the volume
void
DrawCube()
{
        glLineWidth(2.5);
        glColor3f(1.0,1.0,1.0);
        glBegin(GL_LINES);
        
        for (int i = 0; i < 24; i++)
        {
                glVertex3f(vertexDrw[lines[i]][0],
                           vertexDrw[lines[i]][1],
                           vertexDrw[lines[i]][2]);     
        }
        glEnd();
}

//Draws a plane in the Z direction using the texture input texID
void
DrawTexturePlane()
{
	glEnable(GL_TEXTURE_3D);
	glBindTexture(GL_TEXTURE_3D, texID);
	glBegin(GL_QUADS);
		glTexCoord3f(
			z,
			z,
			curPos/::c
			);
		glVertex3f(
			z,
			z,
			curPos
			);
		glTexCoord3f(
			z,
			1.0,
			curPos/::c
			);
		glVertex3f(
			z,
			b,
			curPos
			);
		glTexCoord3f(
			1.0,
			1.0,
			curPos/::c
			);
		glVertex3f(
			a,
			b,
			curPos
			);
		glTexCoord3f(
			1.0,
			z,
			curPos/::c
			);
		glVertex3f(
			a,
			z,
			curPos
			);
		glEnd();
		glBindTexture(GL_TEXTURE_3D, 0);
		glDisable(GL_TEXTURE_3D);
}	

//Either loads the geometry from the trace (once complete)
//Or from file(if the trace is cancelled).
void
CreateList(bool fromSelf)
{
	if(fromSelf)
		net.createFromSelf();
	else
		skeleton.createFromFile(sfname);
}

//glut render method
void
renderScene()
{	
	glDisable(GL_CULL_FACE);
	glEnable(GL_DEPTH_TEST);
	glViewport(0,0, w_size[0], w_size[1]);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
  	glOrtho(0.0, GL[0], 0.0, GL[1], 0.0, GL[2]);
	glMatrixMode(GL_TEXTURE);
	glLoadIdentity();
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glClearColor(0,0,0,0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glTranslatef(GL[0]/2, GL[1]/2, 0.0);
	glScalef(6.0, 6.0, 6.0);
	cam.LookAt((::a)/2.0, (::b)/2.0, (::c)/2.0);
	p	= cam.getPosition();
	up	= cam.getUp();
	d	= cam.getLookAt();
	gluLookAt(p[0], p[1], p[2], d[0], d[1], d[2], up[0], up[1], up[2]);
	CHECK_OPENGL_ERROR
	DrawCube();
	if(cyl == 0)
		net.Render();
	else if(cyl == 1)
		net.RenderCylinders();
	else
		net.RenderAll();
	DrawTexturePlane();
	CHECK_OPENGL_ERROR
	glutSwapBuffers();
}	


//--------------------------RUN TIME/TRACING------------------------//
//------------------------------------------------------------------//

//Runs the trace based on the seedpoints loaded in the Init.
//Stores the result in filename.
void
run(float min_cost, std::string filename)
{
	spidey.trace(min_cost);
	net = spidey.getGLNetwork();
	spidey.saveNetwork(filename);
}

//--------------------CONTROL AND VISUALIZATION---------------------//
//------------------------------------------------------------------//

//change size
void
changeSize(int w, int h)
{
	glViewport(0,0,w,h);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0.0, GL[0], 0.0, GL[1], 0.0, GL[2]);
	glMatrixMode(GL_MODELVIEW);
}
	
//Process mouse button input
void
MouseButton(int button, int state, int x, int y)
{

        if (button == GLUT_LEFT_BUTTON)
        {
                button1          = (state == GLUT_DOWN) ? true : false;
               // button_shift     = glutGetModifiers();
                prevmousePos[0]  = (float)x;
                prevmousePos[1]  = (float)y; 
        }
}
//Process mouse input
void
MouseMotion(int x, int y)
{
        if(button1)
                {       
                        mousePos[0] = (prevmousePos[0] - (float) x)*0.00005;
                        mousePos[1] = ((float)y - prevmousePos[1])*0.00005;
                        prevmousePos[0] = (float)x;
                        prevmousePos[1] = (float)y;
                        cam.OrbitFocus(mousePos[0]*degtorad, mousePos[1]*degtorad);
                }
}

//Process Arrow Key presses.
void
processSpecialKeys(int key, int xx, int yy)
{
	switch(key)
	{
		case GLUT_KEY_UP:
			curPos = curPos + 1.0;		
			break;
		case GLUT_KEY_DOWN:
			curPos = curPos - 1.0;		
			break;
	}
	glutPostRedisplay();
}

//process keys.
void
processKeys(unsigned char key, int x, int y)
{

	if (key == 27)
		exit(0);
	///space
	if (key == 32)
	{
		cyl++;
		if(cyl > 3)
			cyl = 0;
	}
}

//Redraw
void
idleFunction()
{
	glutPostRedisplay();
}

void output_usage(){
	std::cout<<std::endl<<"Usage: volumespider --options [value(s)]" << std::endl;
	std::cout<<std::endl<<"Example usage:" << std::endl;
	std::cout<<"./volumespider --data /home/pavel/Documents/Test_Data/ --voxelsize 0.6 0.6 1.0 --seeds /Documents/VS/volume-spider/mainSeeds.txt --cost 195 --numtemplates 250 125 50 --gui" << std::endl;
}

//-------------------------------MAIN-------------------------------//
//------------------------------------------------------------------//

int
main(int argc, char **argv)
{
	std::vector<std::string>::iterator j;
	int cost = 61;
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(w_size[0], w_size[1]);

	glutCreateWindow("Full_Run_From_Seed_Points");

	GLenum err = glewInit();

	//glut commands for rendering and control.
	glutKeyboardFunc(processKeys);
	glutDisplayFunc(renderScene);
	glutReshapeFunc(changeSize);
	glutMouseFunc(MouseButton);
	glutMotionFunc(MouseMotion);
	glutIdleFunc(idleFunction);	
	glutSpecialFunc(processSpecialKeys);


	stim::arglist args;

#ifdef _WIN32
	args.set_ansi(false);
#endif
	args.add("help", "Prints the usage information");
	args.add("load", "The file path to the data file (.obj) to load for visualization as a string", "", "[string]");
	args.add("seeds", "The file path to the seeds file in .txt format as a string", "./seeds.txt", "[string");
	args.add("savefile", "The name of the save file that will be generated after execution", "network.txt", "[string]");
	args.add("voxelsize", "Three float values representing the size of each voxel in x, y and z. Most likely in microns per voxel", "0.6 0.6 1.0", "[float], [float], [float]");
	args.add("numtemplates", "Three integer values representing the number of direction, position and size templates the algorithm will use for segmentation", "200, 100, 50","[int], [int], [int]");
	args.add("cost", "The cost value to use as a termination criterion for the segmentation","195", "[positive int]");
	args.add("gui", "Run the gui after or before segmentation.");
	
	args.parse(argc, argv);
	
	if(args["help"].is_set()){															//if the user asks for help, provide it
		output_usage();																	//output usage parameters
		std::cout << args.str()<< std::endl;											//output all arguments
		exit(1);																		//end execution
	}
	if(args.nargs() != 1){
		std::cout<<"Enter a path containing the images to be segmented."<<std::endl;
		output_usage();
		std::cout << args.str()<< std::endl;
		exit(1);
	}

	std::vector<std::string> commands;
	std::vector<int> sz;
	sz.resize(3);sz[0] = 512; sz[1] = 512; sz[2] = 228;

	std::vector<float> dm;
	dm.resize(3); dm[0] = 1.0; dm[1] = 1.0; dm[2] = 1.0;
	dm[0] = args["voxelsize"].as_float(0);
	dm[1] = args["voxelsize"].as_float(1);
	dm[2] = args["voxelsize"].as_float(2);

	std::vector<int> templates;
	templates.resize(3);
	templates[0] = args["numtemplates"].as_int(0);
	templates[1] = args["numtemplates"].as_int(1);
	templates[2] = args["numtemplates"].as_int(2);

	spidey.setNumberDirectionTemplates(templates[0]);
	spidey.setNumberPositionTemplates(templates[1]);
	spidey.setNumberSizeTemplates(templates[2]);

	cost = args["cost"].as_int();

	std::string image_path = args.arg(0);

	if (GLEW_OK != err)
	{
		std::cerr << "Failed" << std::endl;
		fprintf(stderr, "Error: %s\n", glewGetErrorString(err));
	}
	std::cout<<"Loading images...";
	stim::gl_texture<unsigned char> stack (image_path + "/*.jpg", GL_LINEAR, GL_CLAMP_TO_EDGE)    ;
    	stack.attach();
	std::cout<<"done."<<std::endl;

	texID = stack.getTexture();
	sz = stack.getSize();
	
	if(args["seeds"].is_set())
	{
		std::string seedName = args["seeds"].as_string();
		spidey.setSeeds(seedName);
	}

	if(args["savefile"].is_set())
	{
		sfname = args["savefile"].as_string();
	}

	if(args["gui"].is_set())
	{
		gui=true;
	}

	

	#ifdef TIMING
		start = std::clock(); 
	#endif
	
	glInit(w_size[0], w_size[1], sz[0], sz[1], sz[2], dm[0], dm[1], dm[2]);
	spidey.trace(cost);
	#ifdef TIMING
		duration = (std::clock() - start) / (double) CLOCKS_PER_SEC;
		std::cerr << "Segmentation Took: " << duration << std::endl;
	#endif
//			spidey.saveNetwork(sfname);
	spidey.saveNetwork("Run_6_7_1024_ii_4_4.obj");
	#ifdef TIMING
		std::vector<double> times = spidey.getTimings();
		std::cout << "Branch detection took " << times[0]/1000 << " seconds" << std::endl;
		std::cout << "Finding optimal direction took " << times[1]/1000 << " seconds" << std::endl;
		std::cout << "Finding optimal position took " << times[2]/1000 << " seconds" << std::endl;
		std::cout << "Finding optimal size took " << times[3]/1000 << " seconds" << std::endl;
		std::cout << "Cost calculation took " << times[4]/1000 << " seconds" << std::endl;
		std::cout << "Network Manipulation took " << times[5]/1000 << " seconds" << std::endl;
		std::cout << "Collision Detection took " << times[6]/1000 << " seconds" << std::endl;

	#endif

	if(gui)
	{
		CHECK_OPENGL_ERROR
		net = spidey.getGLNetwork();
		net.createFromSelf();
		net.createCylinders();
		CHECK_OPENGL_ERROR
		glutMainLoop();
		return 0;
	} else {
		return 0;
	}
}



