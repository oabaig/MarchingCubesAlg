#include "hip/hip_runtime.h"
#include "glnetwork.h"  //must be first
#include <math.h>
#include <cmath>
#include <iostream>
#include <vector>
//#include "stim/math/mathvec.h"
#include <stim/math/vector.h>
//#include <GL/gl.h>
#include <GL/glew.h>
#include <GL/glut.h>
#include <GL/freeglut.h>
//#include <GL/glext.h>
#include "stim/gl/gl_spider.h"
#include "stim/gl/gl_texture.h"
#include "stim/gl/error.h"
#include "stim/visualization/camera.h"
#include <stim/visualization/glObj.h>
#include <stim/math/rect.h>
#include <stim/math/constants.h>

//#include <ANN/ANN.h>
//#include "fiber.h"
#define VERTICAL 	1
#define HORIZONTAL 	1
#define _USE_MATH_DEFINES
		stim::camera cam;
		stim::camera Parker;
		GLuint texID;
		GLuint texID2;
		int id = 0;
//		GLuint fboId;
		stim::vec<float> D(0,0,0);
//		GLuint rboId;
//		GLuint pbo;
		float pvalue = 0.0;
		std::vector<stim::vec<float > > pst;
//		float a = 512.0*0.6;
//		float b = 512.0*0.6;
//		float c = 298.0*1.0;
//		float DIMS[3] = {1024.0, 1024.0, 98.0};
	//	float DIMS[3] = {1024.0, 1024.0, 1024.0};
		float DIMS[3] = {512, 512, 298};
		float VOXDIMS[3] = {0.6, 0.6, 1.0};
		float a = DIMS[0]*VOXDIMS[0];
		float b = DIMS[1]*VOXDIMS[1];
		float c = DIMS[2]*VOXDIMS[2];
		float z = 0.0;
//		float GL[3] = {900.0, 900.0, 40000.0};
		float GL[3] = {1300.0, 1300.0, 40000.0};
//		GLsizei size[2] = {900,900};
		GLsizei size[2] = {1300,1300};
		GLfloat Normals[6][3] =
			{{ -1.0, 0.0, 0.0}, {0.0, 1.0, 0.0}, {1.0, 0.0, 0.0},
			{0.0, -1.0, 0.0}, {0.0, 0.0, 1.0}, {0.0, 0.0, -1.0}};
		GLint faces[6][4] =
			{{0, 1, 2, 3}, {4, 5, 7, 6}, {1, 5, 7, 2},
	 		 {3, 2, 7, 6}, {0, 4, 6, 3}, {0, 1, 5, 4}};
		GLfloat vertex[8][3] =
			{{0.0, 0.0, 1.0}, {1.0, 0.0, 1.0}, {1.0, 1.0, 1.0},
	 		{0.0, 1.0, 1.0}, {0.0, 0.0, 0.0}, {1.0, 0.0, 0.0},
	 		{0.0, 1.0, 0.0}, {1.0, 1.0, 0.0}}; 
		GLint lines[24] = {0,1,1,2,2,3,0,3,4,5,5,7,7,6,6,4,3,6,2,7,1,5,0,4};
		//GLfloat vertexDrw[8][3] =
		//	{{a, a, b}, {b, a, b}, {b, b, b},
	 	//	{a, b, b}, {a, a, a}, {b, a, a},
	 	//	{a, b, a}, {b, b, a}};
		GLfloat vertexDrw[8][3] =
			{{b, b, z}, {z, b, z}, {z, z, z},
	 		{a, z, z}, {a, b, c}, {z, b, c},
	 		{a, z, c}, {z, z, c}};
		static float mousePos[2]        = {0,0};
		static float prevmousePos[2]        = {0,0};
		unsigned long tick 		= 0;
		stim::vec<float> p(0,0,0);
		stim::vec<float> up(0,0,0);
		stim::vec<float> d(0,0,0);
		static bool button1 		= false;
		static bool button_shift	= false;
		static float degtorad		= 360/(stim::TAU);
//		static GLfloat adjustTex	= 1.0/DIMS[2]/2;
//		static GLfloat adjustDrw	= 2.0/DIMS[2]/2;
//		static GLfloat oriTex[3]	= {adjustTex*213+adjustTex/2
//							,adjustTex*213+adjustTex/2
//								,adjustTex*213+adjustTex/2};
//		static GLfloat oriDrw[3]	= {adjustDrw/2
//							,adjustDrw/2
//								,adjustDrw/2};
//		static GLfloat org[2]		= {adjustTex/2,adjustTex/2+425*adjustTex};
		stim::gl_spider<float> spidey(1000, 200, 100);
		stim::vec<float>p1(0,0,0);
		stim::vec<float>p2(0,0,0);
		stim::vec<float>p3(0,0,0);
		stim::vec<float>p4(0,0,0);
		stim::glObj<float>skeleton;
		std::vector<stim::vec3<float> > 	sl;
		std::vector<stim::vec3<float> > 	sv;
		std::vector<float >	       	sm;
		static bool tracingLine 	= false;
		ofstream branches;
//		static bool bfound		= false;
		static bool minimize		= false;
		static bool artificial		= false;


//network stuff
		stim::glnetwork<float> net;
		std::clock_t start;
		double duration;

		std::vector<stim::vec3<float> > Ps;
		std::vector<stim::vec<float> > Ms;
		

void 
glInit()
{
	glEnable(GL_TEXTURE_3D);
	glEnable(GL_DEPTH_TEST);
	glTexEnvi(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glColor4d(0.0, 0.0, 0.0, 1.0);
	glShadeModel(GL_FLAT);
//	cam.setPosition(500.0,500.0,500.0);
	cam.setPosition(1000.0,1000.0,1000.0);
	cam.setFocalDistance(100.0);
	cam.LookAt(300.0, 300.0, 300.0);
	p = cam.getPosition();
	up = cam.getUp();
	d = cam.getLookAt();
	glViewport(0, 0, size[0], size[1]);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0.0, GL[0], 0.0, GL[1], 0.0, GL[2]);
	glMatrixMode(GL_MODELVIEW);
//	spidey = stim::gl_spider<float>(0.515372, 0.564174, 0.494553,
//			0.130563, -0.98295, -0.129467,
//			 0.03);
	//spidey = stim::gl_spider<float>(93.0, 273.0, 78.0,
	//		0.0, 0.0, -1.0,
	//		 100.0);
	//spidey = stim::gl_spider<float>();
	//std::cout<< p1 << ":" << p2 << ":" << p3 <<":" << p4 << std::endl;
//spidey = stim::gl_spider<float>
//	(0.0641842,0.0172776,-0.997788,0.537789,0.587935,0.449583,0.0259658);
//	spidey = stim::gl_spider<float>(0.451825, 0.592827, 0.558536,
 //			0.159298, -0.11443, -0.980576,
//			 0.0367658);
	//spidey = stim::gl_spider<float>(0.3075257, 0.630329, 0.496995,
	//				-0.532453, 0.293348, -0.794993,
	//				0.03);
	spidey.attachSpider(texID);
	spidey.setSize(DIMS[0], DIMS[1], DIMS[2]);
	spidey.setDims(VOXDIMS[0],VOXDIMS[1], VOXDIMS[2]);
	Parker.setPosition(spidey.getPosition());
	Parker.LookAt(spidey.getDirection());
//	GenerateFBO(400, 200);
	spidey.initCuda();
//	std::ifstream myfile("SeedBranchVec.txt");
	std::ifstream myfile("mainSeeds.txt");
//	std::ifstream myfile("SeedBranchVec.txt");
	string line;
	if(myfile.is_open())
	{
		while (getline(myfile, line))
		{
			float x, y, z, u, v, w, m;
			//myfile >> x >> y;
			myfile >> x >> y >> z >> u >> v >> w >> m;
			sl.push_back(stim::vec3<float>(x, y, z));
			sv.push_back(stim::vec3<float>(u,v,w));
			sm.push_back(m);
		}
		myfile.close();
	} else { std::cerr<<"failed" << std::endl;}
//	skeleton = stim::obj<float>("Skeleton.obj");	
	branches.open("branches4.txt");
		pst.resize(8);
		for(int i = 0; i < pst.size(); i++)
			pst[i] = stim::vec<float>(0,0,0);

//	spidey.setPosition(78, 207, 178);
//	spidey.setDirection(-0.073569, 0.484641, 0.871614);
//	spidey.setMagnitude(14.0);

//starting
//	spidey.setPosition(150, 215, 210);
//	spidey.setDirection(0.0214, -0.0657, 0.995531);
//	spidey.setMagnitude(16.864);

//Branch
	spidey.setPosition(0.0, 0.0, 0.0);
//	spidey.setPosition(203.876, 148.344, 245.522);
	spidey.setDirection(0.0, 0.0, 1.0);
	spidey.setMagnitude(16);
	net = spidey.getGLNetwork();
	net.createFromSelf();
	net.createCylinders();
//	skeleton.Begin(stim::OBJ_LINE);

	//glGenRenderbuffers(1, &rboId);
	//glBindRenderbuffer(GL_RENDERBUFFER, rboId);
	//GLuint rboId;
	//glGenRenderbuffers(1, &rboId);
	//glBindRenderbuffer(GL_RENDERBUFFER, rboId);
	//glGenBuffers(1, &pbo);
	//glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
	//glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, 426*426*sizeof(uchar4), NULL, GL_DYNAMIC_DRAW_ARB); 
	CHECK_OPENGL_ERROR
}


void
DrawCube()
{
	glLineWidth(2.5);
	glColor3f(1.0,0.0,0.0);
	glBegin(GL_LINES);
	
	for (int i = 0; i < 24; i++)
	{
		glVertex3f(vertexDrw[lines[i]][0],
			   vertexDrw[lines[i]][1],
			   vertexDrw[lines[i]][2]);	
	}
	glEnd();
}

void
DrawCylinder(float r)
{			  
//			  stim::vec<float> mg = spidey.getMagnitude();
//                        float z0 = -r; float z1 = r; float r0 = r;
                          float z0 = -0.5; float z1 = 0.5; float r0 = 0.5;
                          float x,y;
//                        float xold = 0.0; float yold = 0.0;
                          float xold = 0.5; float yold = 0.0;
                          float step = 360.0/1089.0*36;
                          glEnable(GL_TEXTURE_3D);
                          glBindTexture(GL_TEXTURE_3D, texID);
                          glBegin(GL_QUAD_STRIP);
                                 for(float i = step; i <= 360.0; i += step)
                                 {
                                          x=r0*cos(i*stim::TAU/360.0);
                                          y=r0*sin(i*stim::TAU/360.0);
                                          glTexCoord3f(x,y,z0); 
//                                        glVertex2f(0.0, j*8.0+8.0); 
                                          glVertex3f(x,y,z0);
                                          glTexCoord3f(x,y,z1); 
//                                        glVertex2f(16.0, j*8.0+8.0);
                                          glVertex3f(x, y, z1);
                                          glTexCoord3f(xold,yold,z1); 
//                                        glVertex2f(16.0, j*8.0);
                                          glVertex3f(xold, yold, z1);
                                          glTexCoord3f(xold,yold,z0); 
//                                        glVertex2f(0.0, j*8.0);
                                          glVertex3f(xold, yold, z0);
                                          xold=x;
                                          yold=y;
                                 }                                                                 
                          glEnd();  
}

void
DrawCylinder(std::vector<stim::vec3<float> > inP, std::vector<stim::vec<float> > inM, int withTex = 1)
{			  

	stim::cylinder<float> cyl(inP, inM);
	std::vector<std::vector<stim::vec3<float> > > p = cyl.getPoints(8);
	if(withTex){
        glEnable(GL_TEXTURE_3D);
        glBindTexture(GL_TEXTURE_3D, texID);
	for(int i = 0; i < p.size()-1; i++)
	{
			for(int j = 0; j < p[0].size()-1; j++)
			{
//				glColor4f(1.0, 1.0, 0.0, 0.5);
//				glEnable(GL_BLEND);
//				glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
				glBegin(GL_QUADS);
					glTexCoord3f(p[i][j][0], p[i][j][1], p[i][j][2]);
					glVertex3f(p[i][j][0], p[i][j][1], p[i][j][2]);

					glTexCoord3f(p[i][j+1][0], p[i][j+1][1], p[i][j+1][2]);
					glVertex3f(p[i][j+1][0], p[i][j+1][1], p[i][j+1][2]);

					glTexCoord3f(p[i+1][j+1][0], p[i+1][j+1][1], p[i+1][j+1][2]     );
					glVertex3f(p[i+1][j+1][0], p[i+1][j+1][1], p[i+1][j+1][2]     );

					glTexCoord3f(p[i+1][j][0], p[i+1][j][1], p[i+1][j][2]);
					glVertex3f(p[i+1][j][0], p[i+1][j][1], p[i+1][j][2]);
				glEnd();
//				glDisable(GL_BLEND);

//				glColor4f(1.0, 0.0, 1.0, 1.0);
//				glBegin(GL_LINES);
//					glVertex3f(p[i][j][0], p[i][j][1], p[i][j][2]);
//					glVertex3f(p[i][j+1][0], p[i][j+1][1], p[i][j+1][2]);
//					glVertex3f(p[i+1][j][0], p[i+1][j][1], p[i+1][j][2]);
//					glVertex3f(p[i+1][j+1][0], p[i+1][j+1][1], p[i+1][j+1][2]     );
//				glEnd();
			}
//	glDisable(GL_TEXTURE_3D);

	}

		pst[0] = cyl.surf(pvalue, 45.0);
		pst[1] = cyl.surf(pvalue, 90.0);
		pst[2] = cyl.surf(pvalue, 135.0);
		pst[3] = cyl.surf(pvalue, 180.0);
		pst[4] = cyl.surf(pvalue, 225.0);
		pst[5] = cyl.surf(pvalue, 270.0);
		pst[6] = cyl.surf(pvalue, 315.0);
		pst[7] = cyl.surf(pvalue, 360.0);
	
	} else {
	for(int i = 0; i < p.size()-1; i++)
	{
			for(int j = 0; j < p[0].size()-1; j++)
			{
//				glColor4f(1.0, 1.0, 0.0, 0.5);
//				glEnable(GL_BLEND);
//				glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
				glColor4f(1.0, 1.0, 0.0, 0.5);
				glEnable(GL_BLEND);
				if(j == 0)
				{
				glPushMatrix();
					glTranslatef(p[i][j][0], p[i][j][1], p[i][j][2]);
//					rot = spidey.getRotation(dr);
//					glRotatef(rot[0], rot[1], rot[2], rot[3]);
					glScalef(10, 10, 10);
					glColor3f(0.0, 1.0, 0.0);
		//			glutSolidSphere(0.1,10,10);
				glPopMatrix();	
				}
				glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
				glBegin(GL_QUADS);
					glVertex3f(p[i][j][0], p[i][j][1], p[i][j][2]);

					glVertex3f(p[i][j+1][0], p[i][j+1][1], p[i][j+1][2]);

					glVertex3f(p[i+1][j+1][0], p[i+1][j+1][1], p[i+1][j+1][2]     );

					glVertex3f(p[i+1][j][0], p[i+1][j][1], p[i+1][j][2]);
				glEnd();
				glDisable(GL_BLEND);

				glColor4f(1.0, 0.0, 1.0, 1.0);                                                                                                                                   
				glLineWidth(2.0);
				glBegin(GL_LINES);
					glVertex3f(p[i][j][0], p[i][j][1], p[i][j][2]);
					glVertex3f(p[i][j+1][0], p[i][j+1][1], p[i][j+1][2    ]);
					glVertex3f(p[i][j][0], p[i][j][1], p[i][j][2]);
					glVertex3f(p[i+1][j][0], p[i+1][j][1], p[i+1][j][2    ]     );
				glEnd();

//				glColor4f(1.0, 0.0, 1.0, 1.0);
//				glBegin(GL_LINES);
//					glVertex3f(p[i][j][0], p[i][j][1], p[i][j][2]);
//					glVertex3f(p[i][j+1][0], p[i][j+1][1], p[i][j+1][2]);
//					glVertex3f(p[i+1][j][0], p[i+1][j][1], p[i+1][j][2]);
//					glVertex3f(p[i+1][j+1][0], p[i+1][j+1][1], p[i+1][j+1][2]     );
//				glEnd();
			}
//	glDisable(GL_TEXTURE_3D);

	}
//		cyl.print(0);
//		cyl.print(1);
//		cyl.print(2);
//		cyl.print(3);
//		cyl.print(4);

		glPushMatrix();
//			stim::vec<float> ps;
//			ps = cyl.surf(0.5, 90.0);
			stim::vec3<float> ps = cyl.surf(0.5, 90.0);
			glTranslatef(ps[0], ps[1], ps[2]);
			glScalef(10.0, 10.0, 10.0);
			glColor3f(1.0, 0.0, 0.0);
			glutSolidSphere(0.1,10,10);
		glPopMatrix();	

		glPushMatrix();
			ps = cyl.surf(1.0, 90.0);
			std::cout << ps << std::endl;
			glTranslatef(ps[0], ps[1], ps[2]);
			glScalef(10.0, 10.0, 10.0);
			glColor3f(1.0, 0.0, 0.0);
			glutSolidSphere(0.1,10,10);
		glPopMatrix();	

		glPushMatrix();
			ps = cyl.surf(pvalue, 45.0);
			glTranslatef(ps[0], ps[1], ps[2]);
			glScalef(10.0, 10.0, 10.0);
			glColor3f(1.0, 0.0, 0.0);
			glutSolidSphere(0.1,10,10);
		glPopMatrix();	

		glPushMatrix();
			ps = cyl.surf(pvalue, 90.0);
			glTranslatef(ps[0], ps[1], ps[2]);
			glScalef(10.0, 10.0, 10.0);
			glColor3f(1.0, 0.0, 0.0);
			glutSolidSphere(0.1,10,10);
		glPopMatrix();	

		glPushMatrix();
			ps = cyl.surf(pvalue, 135.0);
			glTranslatef(ps[0], ps[1], ps[2]);
			glScalef(10.0, 10.0, 10.0);
			glColor3f(1.0, 0.0, 0.0);
			glutSolidSphere(0.1,10,10);
		glPopMatrix();	

		glPushMatrix();
			ps = cyl.surf(pvalue, 180.0);
			glTranslatef(ps[0], ps[1], ps[2]);
			glScalef(10.0, 10.0, 10.0);
			glColor3f(1.0, 0.0, 0.0);
			glutSolidSphere(0.1,10,10);
		glPopMatrix();	

		glPushMatrix();
			ps = cyl.surf(pvalue, 225.0);
			glTranslatef(ps[0], ps[1], ps[2]);
			glScalef(10.0, 10.0, 10.0);
			glColor3f(1.0, 0.0, 0.0);
			glutSolidSphere(0.1,10,10);
		glPopMatrix();	

		glPushMatrix();
			ps = cyl.surf(pvalue, 270.0);
			glTranslatef(ps[0], ps[1], ps[2]);
			glScalef(10.0, 10.0, 10.0);
			glColor3f(1.0, 0.0, 0.0);
			glutSolidSphere(0.1,10,10);
		glPopMatrix();	

		glPushMatrix();
			ps = cyl.surf(pvalue, 315.0);
			glTranslatef(ps[0], ps[1], ps[2]);
			glScalef(10.0, 10.0, 10.0);
			glColor3f(1.0, 0.0, 0.0);
			glutSolidSphere(0.1,10,10);
		glPopMatrix();	

		glPushMatrix();
			ps = cyl.surf(pvalue, 360.0);
			glTranslatef(ps[0], ps[1], ps[2]);
			glScalef(10.0, 10.0, 10.0);
			glColor3f(1.0, 0.0, 0.0);
			glutSolidSphere(0.1,10,10);
		glPopMatrix();	
	}



}


void
DrawSphere()
{
	int vang;
	int ang;
	int delang = 5;
	float r0 = 0.2;
	float x0, y0, z0, x1, y1, z1, x2, z2;
	
	glEnable(GL_TEXTURE_3D);
	glBindTexture(GL_TEXTURE_3D, texID);
	glBegin(GL_QUADS);
	for (vang = 0; vang <= 180; vang+=delang){
		y0=r0*cos((double)(vang)*stim::TAU/360.0);
     		y1=r0*cos((double)(vang+delang)*stim::TAU/360.0);
     		x0=r0*sin((double)vang*stim::TAU/360.0);
     		z0=0.0; 	
		for (ang=0;ang<=360;ang+=delang)
		     {
			x1=r0*cos((double)ang*stim::TAU/360.0)*sin((double)vang*stim::TAU/360.0);
			x2=r0*cos((double)ang*stim::TAU/360.0)*sin((double)(vang+delang)*stim::TAU/360.0);
			z1=r0*sin((double)ang*stim::TAU/360.0)*sin((double)vang*stim::TAU/360.0);
			z2=r0*sin((double)ang*stim::TAU/360.0)*sin((double)(vang+delang)*stim::TAU/360.0);
	//		glTexCoord3f(x0,y0,z0);
			glVertex3f(x0,y0,z0);
	//		glTexCoord3f(x1,y0,z1);
			glVertex3f(x1,y0,z1);
	//		glTexCoord3f(x1,y0,z1);
			glVertex3f(x1,y0,z1);
	//		glTexCoord3f(x2,y1,z2);
			glVertex3f(x2,y1,z2);
			x0=x1;
			z0=z1;
		     }
		  }
     glEnd(); 
}

	
void
DrawPlanes()
{
	glEnable(GL_TEXTURE_3D);
	glBindTexture(GL_TEXTURE_3D, texID);
	p1 = spidey.ver.p(1,1);
	p2 = spidey.ver.p(1,0);
	p3 = spidey.ver.p(0,0);
	p4 = spidey.ver.p(0,1);
	//glPushName(VERTICAL);
	glBegin(GL_QUADS);
		glTexCoord3f(
			p1[0]/a,
			p1[1]/b,
			p1[2]/c
			);
		glVertex3f(
			p1[0],
			p1[1],
			p1[2]
			);
		glTexCoord3f(
			p2[0]/a,
			p2[1]/b,
			p2[2]/c
			);
		glVertex3f(
			p2[0],
			p2[1],
			p2[2]
			);
		glTexCoord3f(
			p3[0]/a,
			p3[1]/b,
			p3[2]/c
			);
		glVertex3f(
			p3[0],
			p3[1],
			p3[2]
			);
		glTexCoord3f(
			p4[0]/a,
			p4[1]/b,
			p4[2]/c
			);
		glVertex3f(
			p4[0],
			p4[1],
			p4[2]
			);
	glEnd();
	//glPopName();
	p1 = spidey.hor.p(1,1);
	p2 = spidey.hor.p(1,0);
	p3 = spidey.hor.p(0,0);
	p4 = spidey.hor.p(0,1);
	//glPushName(HORIZONTAL);
	glBegin(GL_QUADS);
		glTexCoord3f(
			p1[0]/a,
			p1[1]/b,
			p1[2]/c
			);
		glVertex3f(
			p1[0],
			p1[1],
			p1[2]
			);
		glTexCoord3f(
			p2[0]/a,
			p2[1]/b,
			p2[2]/c
			);
		glVertex3f(
			p2[0],
			p2[1],
			p2[2]
			);
		glTexCoord3f(
			p3[0]/a,
			p3[1]/b,
			p3[2]/c
			);
		glVertex3f(
			p3[0],
			p3[1],
			p3[2]
			);
		glTexCoord3f(
			p4[0]/a,
			p4[1]/b,
			p4[2]/c
			);
		glVertex3f(
			p4[0],
			p4[1],
			p4[2]
			);
	glEnd();
	//glPopName();
	glBindTexture(GL_TEXTURE_3D, 0);
	glDisable(GL_TEXTURE_3D);
}
void
DrawSpiders()
{
	glEnable(GL_TEXTURE_3D);
	glBindTexture(GL_TEXTURE_3D, texID);
	p1 = spidey.hor.p(1,1);
	p2 = spidey.hor.p(1,0);
	p3 = spidey.hor.p(0,0);
	p4 = spidey.hor.p(0,1);
	glBegin(GL_QUADS);
		glTexCoord3f(
			p1[0]/a,
			p1[1]/b,
			p1[2]/c
			);
		glVertex2f(0.0,0.0);
		glTexCoord3f(
			p2[0]/a,
			p2[1]/b,
			p2[2]/c
			);
		glVertex2f(1.0, 0.0);
		glTexCoord3f(
			p3[0]/a,
			p3[1]/b,
			p3[2]/c
			);
		glVertex2f(1.0, 2.0);
		glTexCoord3f(
			p4[0]/a,
			p4[1]/b,
			p4[2]/c
			);
		glVertex2f(0.0, 2.0);
	glEnd();
	p1 = spidey.ver.p(1,1);
	p2 = spidey.ver.p(1,0);
	p3 = spidey.ver.p(0,0);
	p4 = spidey.ver.p(0,1);
	
	glBegin(GL_QUADS);
		glTexCoord3f(
			p1[0]/a,
			p1[1]/b,
			p1[2]/c
			);
		glVertex2f(1.0, 0.0);
		glTexCoord3f(
			p2[0]/a,
			p2[1]/b,
			p2[2]/c
			);
		glVertex2f(2.0, 0.0);
		glTexCoord3f(
			p3[0]/a,
			p3[1]/b,
			p3[2]/c
			);
		glVertex2f(2.0, 2.0);
		glTexCoord3f(
			p4[0]/a,
			p4[1]/b,
			p4[2]/c
			);
		glVertex2f(1.0, 2.0);
	glEnd();
	glBindTexture(GL_TEXTURE_3D, 0);
	glDisable(GL_TEXTURE_3D);
	p1 = spidey.hor.p(1,1);
	p2 = spidey.hor.p(1,0);
	p3 = spidey.hor.p(0,0);
	p4 = spidey.hor.p(0,1);
	}

//render cube	

void
artificialCylinder()
{
	DrawCylinder(Ps, Ms, 0);
}

void
renderScene()
{	
	glViewport(0, 0, size[0], size[1]);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0.0, GL[0],0.0, GL[1], 0.0, GL[2]);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glMatrixMode(GL_TEXTURE);          //these lines must be added in order to make the correct viewing of the texture matrix when getSample is called in gl_spider
	glLoadIdentity();
	glMatrixMode(GL_MODELVIEW);
	glInitNames();
	glClearColor(0,0,0,0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(450.0,450.0,0.0);
	glScalef(4.0, 4.0, 4.0);
	if(!artificial){
		stim::vec3<float> pos = spidey.getPosition();
		cam.LookAt(pos[0], pos[1], pos[2]);
		p = cam.getPosition();
		up = cam.getUp();
		d = cam.getLookAt();
		gluLookAt(p[0], p[1], p[2], d[0], d[1], d[2], up[0], up[1], up[2]);  
	} else {
		cam.LookAt(Ps[0][0], Ps[0][1], Ps[0][2]);
		p = cam.getPosition();
		up = cam.getUp();
		d = cam.getLookAt();
		gluLookAt(p[0], p[1], p[2], d[0], d[1], d[2], up[0], up[1], up[2]);  
	}
	DrawCube();
	//glTranslatef(78.0,273.0, 93.0);
	DrawPlanes();
	stim::vec3<float> ps = spidey.getPosition();
	stim::vec3<float> dr = spidey.getDirection();
	stim::vec3<float> mg = spidey.getMagnitude();
	stim::vec<float> rot = spidey.getRotation(dr);
	glMatrixMode(GL_TEXTURE);
	glPushMatrix();
		glScalef(1.0/a, 1.0/b, 1.0/c);	
		glTranslatef(ps[0], ps[1], ps[2]);
		glRotatef(rot[0], rot[1], rot[2], rot[3]);
		glScalef(mg[0], mg[0], mg[0]);
	glMatrixMode(GL_MODELVIEW);
	glPushMatrix();
		glTranslatef(ps[0], ps[1], ps[2]);
		glRotatef(rot[0], rot[1], rot[2], rot[3]);
		glScalef(mg[0], mg[0], mg[0]);
		DrawCylinder(14);
	glPopMatrix();
	glMatrixMode(GL_TEXTURE);
	glPopMatrix();
	glMatrixMode(GL_MODELVIEW);
	std::stack< stim::vec3<float> > sds = spidey.getSeeds();
//	net = spidey.getGLNetwork();
	for(int i = id; i < sl.size(); i++)
	{
		ps = sl[i];
		dr = sv[i];
		glDisable(GL_TEXTURE_2D);
		glDisable(GL_TEXTURE_3D);
		glDisable(GL_LIGHTING);
		glPushMatrix();
			glTranslatef(ps[0], ps[1], ps[2]);
			rot = spidey.getRotation(dr);
			glRotatef(rot[0], rot[1], rot[2], rot[3]);
			glScalef(mg[0], mg[0], mg[0]);
			glColor3f(1.0, 0.0, 0.0);
			glutSolidSphere(0.1,10,10);
		glPopMatrix();	
	}
	if(!minimize){	
		glDisable(GL_TEXTURE_3D);
		for(int i = 0; i < pst.size(); i++){
			glPushMatrix();
				glTranslatef(pst[i][0], pst[i][1], pst[i][2]);
				glScalef(10.0, 10.0, 10.0);
				glColor3f(0.0, 0.125*i, 1.0);
				glutSolidSphere(0.1,10,10);
			glPopMatrix();	
		}
	}
/*	for(int i = 0; i < id; i++)
	{
		ps = sl[i];
		dr = sv[i];
		glDisable(GL_TEXTURE_2D);
		glDisable(GL_TEXTURE_3D);
		glDisable(GL_LIGHTING);
		glPushMatrix();
			glTranslatef(ps[0], ps[1], ps[2]);
			rot = spidey.getRotation(dr);
			glRotatef(rot[0], rot[1], rot[2], rot[3]);
			glScalef(mg[0], mg[0], mg[0]);
			glColor3f(0.0, 1.0, 0.0);
			glutSolidSphere(0.1,10,10);
		glPopMatrix();	
	}
*/
	for(int i = 0; i < net.sizeV(); i++)
	{
		glDisable(GL_TEXTURE_2D);
		glDisable(GL_TEXTURE_3D);
		glDisable(GL_LIGHTING);
		glPushMatrix();
			stim::vec3<float> a = net.V[i].getPosition();
			glTranslatef(a[0], a[1], a[2]);
			rot = spidey.getRotation(dr);
			glRotatef(rot[0], rot[1], rot[2], rot[3]);
			glScalef(mg[0], mg[0], mg[0]);
			glColor3f(0.0, 0.0, 1.0);
			glutSolidSphere(0.1,10,10);

			//label the nodes with numbers.
			float curTrans[16];
			stim::matrix<float, 4> cT;
			glGetFloatv(GL_MODELVIEW_MATRIX, curTrans);
			cT.set(curTrans);
			stim::vec<float> loc(0.0,0.0,0.0);
			loc = cT*loc;
			ostringstream ss;
			ss<<i<<" "<<net.V[i].edges_to_str();// << " "<<net.V[i].str();
			glColor3f(1., 1., 0.0);
			glRasterPos3f(loc[0]+0.2, loc[1]+0.2, loc[2]+0.2);
			glutBitmapString(GLUT_BITMAP_HELVETICA_18,
			 (const unsigned char* )(ss.str().c_str()));
		glPopMatrix();	
	}

	for (int i = 0; i < net.sizeE(); i++)
	{
		glDisable(GL_TEXTURE_2D);
		glDisable(GL_TEXTURE_3D);
		glDisable(GL_LIGHTING);
		glPushMatrix();
			std::vector<stim::vec3<float> > tmp = net.getEdgeCenterLine(i);
			stim::vec3<float> a = tmp[tmp.size()/2];
			glTranslatef(a[0], a[1], a[2]);
			rot = spidey.getRotation(dr);
			glRotatef(rot[0], rot[1], rot[2], rot[3]);
			glScalef(mg[0], mg[0], mg[0]);
			glColor3f(1.0, 0.0, 1.0);

			float curTrans[16];
			stim::matrix<float, 4> cT;
			glGetFloatv(GL_MODELVIEW_MATRIX, curTrans);
			cT.set(curTrans);
			stim::vec<float> loc(0., 0., 0.);
			loc = cT*loc;
			ostringstream ss;
			ss<<i; // << net.nodes_to_str(i);
			glRasterPos3f(loc[0]+0.2, loc[1]+0.2, loc[2]+0.2);
			glutBitmapString(GLUT_BITMAP_HELVETICA_18, 
				(const unsigned char*)(ss.str().c_str()));
		glPopMatrix();

		glPushMatrix();
			glMatrixMode(GL_PROJECTION);
			glLoadIdentity();
			glOrtho(0.0, GL[0],0.0, GL[1], 0.0, GL[2]);
			glMatrixMode(GL_MODELVIEW);
			glLoadIdentity(); 
			glTranslatef(450.0,450.0,0.0);
			glScalef(4.0, 4.0, 4.0);
			stim::vec3<float> pos = spidey.getPosition();
			cam.LookAt(pos[0], pos[1], pos[2]);
			p = cam.getPosition();
			up = cam.getUp();
			d = cam.getLookAt();
			gluLookAt(p[0], p[1], p[2], d[0], d[1], d[2], up[0], up[1], up[2]);  
			glMatrixMode(GL_TEXTURE);
			glLoadIdentity();
			glScalef(1.0/::a, 1.0/b, 1.0/c); 
			if(!minimize && (i == net.sizeE()-1))
				DrawCylinder(net.getEdgeCenterLine(i), net.getEdgeCenterLineMag(i));
			CHECK_OPENGL_ERROR
			glDisable(GL_TEXTURE_3D);
			glMatrixMode(GL_MODELVIEW);
		glPopMatrix();
	}
//	net.createFromSelf();
	net.Render();	
	if(!minimize){
		net.RenderCylinders();
	}
	if(artificial){
		artificialCylinder();
	}
//	skeleton.createFromSelf();
//	skeleton.Render();
//	for(int i = 0; i < net.E.size(); i++)
//	{
//		glColor3f(0.15*i, 0.0, 0.0); 
//		skeleton.RenderLine(net.E[i].centerline());
//	}	


	//DrawCylinder();
		//glEnable(GL_TEXTURE_2D);


		//glBindTexture(GL_TEXTURE_2D, texID);
		//glBindFramebuffer(GL_FRAMEBUFFER, fboId);
		//glFramebufferTexture2D(
		//		GL_FRAMEBUFFER,
		//		GL_COLOR_ATTACHMENT0,
		//		GL_TEXTURE_2D,
		//		texID2,
		//		0
		//		);
		//glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, texID2, 0);
		//glBindFramebuffer(GL_FRAMEBUFFER, fboId);
		//GLenum DrawBuffers[1] = {GL_COLOR_ATTACHMENT0};
		//glDrawBuffers(1, DrawBuffers);
		//if(glCheckFramebufferStatus(GL_FRAMEBUFFER) != GL_FRAMEBUFFER_COMPLETE)
		//	std::cout << "damn" << std::endl;
		//glBindTexture(GL_TEXTURE_2D, texID2);
		//glClearColor(1,1,1,1);
	//glClear(GL_COLOR_BUFFER_BIT);
	//glLoadIdentity();
	//glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	CHECK_OPENGL_ERROR
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	if(minimize == false){
		glViewport(0,0,400,200);
		gluOrtho2D(0.0, 2.0, 0.0, 2.0);
	//glTranslatef(2.5,2.5,0.0);
	//glScalef(2.0, 2.0, 2.0);
		DrawSpiders();
	//glPopMatrix();
	//DrawSpiders2();
	//glFlush();
	//glFinish();
		CHECK_OPENGL_ERROR
		glBindFramebuffer(GL_FRAMEBUFFER, 0);
		glBindTexture(GL_TEXTURE_2D, 0);
		glBindFramebuffer(GL_READ_FRAMEBUFFER, spidey.getFB());
		glBindFramebuffer(GL_DRAW_FRAMEBUFFER, 0);
		glBlitFramebuffer(0,0,16,218, 0, 0, 16, 218, GL_COLOR_BUFFER_BIT, GL_NEAREST);
	}
	CHECK_OPENGL_ERROR
	//glBindFramebuffer(GL_FRAMEBUFFER,0);
	//glBindTexture(GL_TEXTURE_3D, texID);
	//glGenerateMipmap(GL_TEXTURE_3D);
	//glBindTexture(GL_TEXTURE_3D, 0);
	//glViewport(0,0, 800,800);

	//GLint curbuf;
	//glGetIntegerv(GL_ARRAY_BUFFER_BINDING, &curbuf); 
	//std::cout << curbuf << std::endl;

	//glViewport(0,0,600,600);
	//glBindFramebuffer(GL_FRAMEBUFFER, fboId);
	
	//glDrawBuffer(GL_FRAMEBUFFER0);
	//glDrawBuffer(fboId);
	//glClearColor(1,1,1,1);
	//glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	//glLoadIdentity();
	//std::cout << fboId << std::endl;
	//DrawSpiders();
	
	//glBlitFramebuffer(0.0,0.0,1,1,0.0,0.0,1,1, GL_COLOR_BUFFER_BIT, GL_LINEAR);
	//gluOrtho2D(-5.0, -3.0, -5.0, -1.0);
	//glClearColor(1,1,1,1);
	//glBindTexture(GL_TEXTURE_3D, texID);
	//glPushMatrix();
	//glGenerateMipmap(GL_TEXTURE_3D);
	//glBindTexture(GL_TEXTURE_3D, 0);
	//glGetIntegerv(GL_ARRAY_BUFFER_BINDING, &curbuf); 
	//std::cout << curbuf << std::endl;

//	glCopyImageSubData(fboId, GL_FRAMEBUFFER, 0, 0, 0, 0, GL_FRONT_AND_BACK, GL_NONE, 0, 0, 0, 0, 20, 20, 0);
	
	
	glutSwapBuffers();
}


void
MouseButton(int button, int state, int x, int y)
{

	if (button == GLUT_LEFT_BUTTON)
	{
	 	button1 	 = (state == GLUT_DOWN) ? true : false;
		button_shift     = glutGetModifiers();
		prevmousePos[0]	 = (float)x;
		prevmousePos[1]	 = (float)y; 
	}
	if (button == GLUT_RIGHT_BUTTON && state == GLUT_DOWN)
	{
		branches << "Branch" << "\n";
		branches << spidey.getPosition() << "\n"
			<< spidey.getDirection() << "\n"
			<< spidey.getMagnitude() << "\n";
		branches << "\n";
		
		
	}
}

void MouseMotion(int x, int y)
{
	if(button1 && !button_shift)
		{	
			mousePos[0] = (prevmousePos[0] - (float) x)*0.00005;
			mousePos[1] = ((float)y - prevmousePos[1])*0.00005;
			prevmousePos[0] = (float)x;
			prevmousePos[1] = (float)y;
			cam.OrbitFocus(mousePos[0]*degtorad, mousePos[1]*degtorad);
		}
	if(button1 && button_shift)
		{
			mousePos[0] = (prevmousePos[0] - (float) x)*0.00001;
			mousePos[1] = ((float)y - prevmousePos[1])*0.00001;
			prevmousePos[0] = (float)x;
			prevmousePos[1] = (float)y;
			Parker.Pan(mousePos[0]*degtorad);
			Parker.Tilt(mousePos[1]*degtorad);
			spidey.setDirection(Parker.getDirection());
			spidey.Update();
			std::cout << Parker.getLookAt() << std::endl;
		}
}

void
idleFunction()
{
	tick += 10;
	glutPostRedisplay();
}

void
printCost()
{
	
}


void
processSpecialKeys(int key, int xx, int yy)
{
switch(key) {
	case GLUT_KEY_UP:
//	    oriTex[2] = oriTex[2]+adjustTex;
//	    if (oriTex[2] > org[1]){
//		oriTex[2] = org[1];
//	    }
//	    else{
//		oriDrw[2] = oriDrw[2] + adjustDrw;
//	    }
		pvalue += 0.001;
		std::cerr << pvalue << std::endl;
	    if (glutGetModifiers() == GLUT_ACTIVE_SHIFT)
		{
			stim::vec3<float> temp =	spidey.getPosition();
			temp[1] += 1;
			spidey.setPosition(temp);
			spidey.Update();
		} 
	    if (glutGetModifiers() == GLUT_ACTIVE_CTRL)
		{
			//stim::vec<float> temp = Parker.getDirection();
			//temp[1] += 0.001;
			Parker.Pan(0.01);
			spidey.setDirection(Parker.getDirection());
			spidey.Update();
		}
	    break;
	case GLUT_KEY_DOWN:
//	    oriTex[2] = oriTex[2]-adjustTex;
//	    if (oriTex[2] < org[0]){
//              oriTex[2] = org[0];
//	    }
//	    else{
//		oriDrw[2] = oriDrw[2] - adjustDrw;
//	    }
		pvalue -= 0.01;
		std::cerr << pvalue << std::endl;
	    if (glutGetModifiers() == GLUT_ACTIVE_SHIFT)
		{
			stim::vec3<float> temp =	spidey.getPosition();
			temp[1] -= 1;
			spidey.setPosition(temp);
			spidey.Update();
		}
	    if (glutGetModifiers() == GLUT_ACTIVE_CTRL)
		{
			//stim::vec<float> temp = Parker.getDirection();
			//temp[1] -= 0.001;
			Parker.Pan(0.01);
			spidey.setDirection(Parker.getDirection());
			spidey.Update();
		}
	    break;
	case GLUT_KEY_LEFT:
//	    oriTex[1] = oriTex[1]+adjustTex;
//	    if (oriTex[1] > org[1]){
//		oriTex[1] = org[1];
//          }
//	    else{
//		oriDrw[1] = oriDrw[1] + adjustDrw;
//	    }
	    if (glutGetModifiers() == GLUT_ACTIVE_SHIFT)
		{
			stim::vec3<float> temp =	spidey.getPosition();
			temp[0] += 1;
			spidey.setPosition(temp);
			spidey.Update();
		} 
	    if (glutGetModifiers() == GLUT_ACTIVE_CTRL)
		{
			//stim::vec<float> temp = Parker.getDirection();
			//temp[0] += 0.001;
			Parker.Tilt(0.01);
			spidey.setDirection(Parker.getDirection());
			spidey.Update();
		}
	    break;
	case GLUT_KEY_RIGHT:
//	    oriTex[1] = oriTex[1]-adjustTex;
//	    if (oriTex[1] < org[0]){
//                oriTex[1] = org[0];
//            }
//	    else{
//		oriDrw[1] = oriDrw[1] - adjustDrw;
//	    }
	    if (glutGetModifiers() == GLUT_ACTIVE_SHIFT)
		{
			stim::vec3<float> temp =	spidey.getPosition();
			temp[0] -= 1;
			spidey.setPosition(temp);
			spidey.Update();
		} 
	    if (glutGetModifiers() == GLUT_ACTIVE_CTRL)
		{
			//stim::vec<float> temp = Parker.getDirection();
			//temp[0] -= 0.001;
			Parker.Tilt(-0.01);
			spidey.setDirection(Parker.getDirection());
			spidey.Update();
		}
	    break;
	}	
	glutPostRedisplay();
}


void
QUIT()
{
	skeleton.End();
	skeleton.save("Skeleton.obj");
	stim::glObj<float>skeleton_final;
	for(int i = 0; i < net.sizeE(); i++)
	{
		std::vector<stim::vec< float > > cm = net.getEdgeCenterLineMag(i);
		std::vector<stim::vec3< float > > ce = net.getEdgeCenterLine(i);
		skeleton_final.Begin(stim::OBJ_LINE);
		for(int j = 0; j < ce.size(); j++)
		{
			skeleton.TexCoord(cm[j][0]);
			skeleton.Vertex(ce[j][0], ce[j][1], ce[j][2]);
		}
		skeleton_final.End();
	}
	skeleton.save("Final.obj");
	branches.close();
	exit(0);
}

void
processKeys(unsigned char key, int x, int y)
{
	//ESC key
        if (key == 27)
	{
	     duration = (std::clock() - start)/ (double) CLOCKS_PER_SEC;
	     std::cout << duration << " seconds" << std::endl;
	     QUIT();
        //     exit(0);
	}
	//Space bar
	if (key == 32)
	{
			std::cout <<id<<":"<< sl[id][0] << ", " << sl[id][1]  << ", "  << sl[id][2] << std::endl;
			std::cout <<id<<":"<< sv[id][0] << ", " << sv[id][1]  << ", "  << sv[id][2] << std::endl;
			std::cout <<id<<":"<< sm[id] << std::endl;
		skeleton.Begin(stim::OBJ_LINE);
		tracingLine = true;
//		spidey.Bind();
		spidey.clearCurrent();
		spidey.traceLine(spidey.getPosition(),
			 spidey.getMagnitude(), 200);
//		spidey.printSizes();
//		std::cerr << "got here" << std::endl;
/*		if(tracingLine){
			std::vector<stim::vec<float> > ce = a.first.centerline();
			std::vector<stim::vec<float> > cm = a.first.centerlinemag();
			if(ce.size() > 2)
			{
				for(int i = 0; i < ce.size(); i++)
				{
					skeleton.TexCoord(cm[i][0]);
					skeleton.Vertex(ce[i][0], ce[i][1], ce[i][2]);
				}
				//add edge() to the network.
				if(a.second == -1)
				{
					std::cout << "got here" << std::endl;
					net.addEdge(ce, cm, -1, -1);
				}
				else if(a.second != -1)
				{
					std::cout << "got here1" << std::endl;
					net.addEdge(ce, cm, -1, a.second);
				}
			}
		}	*/
		while(!spidey.Empty()){
			sl.push_back(spidey.getLastSeed());
			sv.push_back(spidey.getLastSeedVec());
			sm.push_back(spidey.getLastSeedMag());
			spidey.popSeed();
		}
//		spidey.Update();	
		skeleton.End();
		tracingLine = false;
		id++;
		spidey.setPosition(sl[id][0], sl[id][1], sl[id][2]);
		spidey.setDirection(sv[id][0], sv[id][1], sv[id][2]);
		spidey.setMagnitude(sm[id]);
		net = spidey.getGLNetwork();
		net.createFromSelf();
		net.createCylinders();
		glutPostRedisplay();
		
//		spidey.setMagnitude(16.0);
//		spidey.Update();
//		std::cout << "stuff1" << std::endl;
			
		
	/*
//		spidey.Bind();
		int cost = spidey.StepP();
		stim::vec<float> m = spidey.getMagnitude();
		stim::vec<float> p = spidey.getPosition();
		if(tracingLine){
			skeleton.TexCoord(m[0]);
			skeleton.Vertex(p[0], p[1], p[2]);
		}
		stim::vec<float> pz = spidey.getPosition();
		spidey.Update();
		std::cerr << "got before" << std::endl;
		while(!spidey.Empty()){
			sl.push_back(spidey.getLastSeed());
			sv.push_back(spidey.getLastSeedVec());
			spidey.popSeed();
		}	
		std::cerr << "got after" << std::endl;
	     	std::cerr << cost << ":" << spidey.getMagnitude()  << ":" << pz << std::endl;
	*/
	}


	//ENTERKEY
	if(key == 13)
	{
		std::cout << "Took a step" << std::endl;
		stim::vec3<float> temp = spidey.getPosition();
		printf("P: %f %f %f \n", temp[0], temp[1], temp[2]);
		temp = spidey.getDirection();
		printf("D: %f %f %f \n", temp[0], temp[1], temp[2]);
		temp = spidey.getMagnitude();
		printf("P: %f \n", temp[0]);
		spidey.Step();
		spidey.Update();
	}

	//m key
	if(key == 109)
	{
		if(minimize == false)
			minimize = true;
		else
			minimize = false;	
	}
	
	// plus key	
	if (key == 43)
	{
		stim::vec<float> temp = spidey.getMagnitude();
		temp = temp + 0.1;
		spidey.setMagnitude(temp);
		spidey.Update();
	}   
	// minus key
	if (key == 45)
	{
		stim::vec<float> temp = spidey.getMagnitude();
		temp = temp - 0.1;
		spidey.setMagnitude(temp);
		spidey.Update();
	}
	if (key == 8)
	{
		//	std::cout <<id<<":"<< sl[id][0] << ", " << sl[id][1]  << ", "  << sl[id][2] << std::endl;
		//	std::cout <<id<<":"<< sv[id][0] << ", " << sv[id][1]  << ", "  << sv[id][2] << std::endl;
		//int cost = spidey.Step();
		spidey.setPosition(sl[id][0], sl[id][1], sl[id][2]);
		spidey.setDirection(sv[id][0], sv[id][1], sv[id][2]);
		spidey.setMagnitude(sm[id]);
//		spidey.setMagnitude(16.0);
		spidey.Update();
		id++;
	}

	if (key == 47)
	{
		if(!tracingLine){
			std::cout << "I have begin a line" << std::endl;
			skeleton.Begin(stim::OBJ_LINE);
			tracingLine = true;
		} else {
			std::cout << "Please End previous line" << std::endl;
		}
	}
	if (key == 92)
	{
		if(tracingLine){
			std::cout << "I have ended a line" << std::endl;
			skeleton.End();
			tracingLine = false;
			
		} else {
			std::cout << "No line being traced, please start a line" << std::endl;
		}
	}
	// "P" key
	if (key == 80)
	{
		net = spidey.getGLNetwork();
		net.to_csv();
		//net.to_gdf();
	}
	// "T" key
	if (key == 84)
	{
		while(id != sl.size())
		{
			skeleton.Begin(stim::OBJ_LINE);
			tracingLine = true;
			spidey.clearCurrent();
			spidey.traceLine(spidey.getPosition(),
				 spidey.getMagnitude(), 200);
//			spidey.printSizes();
			if(id == sl.size()-1)
				break;
			while(!spidey.Empty()){
				sl.push_back(spidey.getLastSeed());
				sv.push_back(spidey.getLastSeedVec());
				sm.push_back(spidey.getLastSeedMag());
				spidey.popSeed();
			}
			spidey.Update();	
			skeleton.End();
			tracingLine = false;
			id++;
			spidey.setPosition(sl[id][0], sl[id][1], sl[id][2]);
			spidey.setDirection(sv[id][0], sv[id][1], sv[id][2]);
			spidey.setMagnitude(sm[id]);
			spidey.Update();
//			std::cout <<" " << std::endl;
//			std::cout << sl.size() << ":" << id << std::endl;
//			std::cout <<" " << std::endl;
		}
		net = spidey.getGLNetwork();
		net.createFromSelf();
		net.createCylinders();
		glutPostRedisplay();
//		net.to_csv();
//		net.to_gdf();
	} 

	///a key
	if (key == 97)
	{
		if(!artificial == true && Ps.size() == 0)
		{		
/*
			Ps.push_back(stim::vec<float>(0.0,0.0,0.0));
			Ps.push_back(stim::vec<float>(0.0,40.0,0.0));
			Ps.push_back(stim::vec<float>(80.0,40.0,0.0));
			Ps.push_back(stim::vec<float>(80.0,0.0,0.0));
			Ms.push_back(stim::vec<float>(10,5));
			Ms.push_back(stim::vec<float>(5,6));
			Ms.push_back(stim::vec<float>(15,6));
			Ms.push_back(stim::vec<float>(5,6)); 
*/
			Ps.push_back(stim::vec<float>(0.0,0.0,0.0));
			Ps.push_back(stim::vec<float>(0.0,40.0,80.0));
			Ps.push_back(stim::vec<float>(40.0,80.0,40.0));
			Ps.push_back(stim::vec<float>(80.0,40.0,20.0));
			Ps.push_back(stim::vec<float>(80.0,0.0,0.0));
			Ms.push_back(stim::vec<float>(10,5));
			Ms.push_back(stim::vec<float>(5,6));
			Ms.push_back(stim::vec<float>(15,6));
			Ms.push_back(stim::vec<float>(5,6));
			Ms.push_back(stim::vec<float>(10,6));

			artificial = true;
		} else if(!artificial == true){
			artificial = true;
		} else {
			artificial = false;
		}
	}
/*	float S[4] = {0.0, 0.6, 0.6, 2.0};
	std::cout << "DEBUG: Main::338" << std::endl;                        
			    std::cout << "pos: " << spidey.getPosition() << std::endl;
			    std::cout << "dir: " << spidey.getDirection() << std::endl;
			    std::cout << "mag: " << spidey.getMagnitude() << std::endl;
			    std::cout << "After Transformation" << std::endl;
			std::cout << "[" << (p1[0])/512/S[1] << ", "
                                   <<(p1[1])/512/S[2] << ", "
                                   <<(p1[2])/426/S[3] << "]" << ":"
                                   << "[" << (p2[0])/512/S[1] << ", " 
                                   <<  (p2[1])/512/S[2] << ", "
                                   <<  (p2[2])/426/S[3] << "]" << ":"
                                   << "[" << (p3[0])/512/S[1] << ", " 
                                   <<  (p3[1])/512/S[2] << ", "
                                   <<  (p3[2])/426/S[3] << "]" << ":"
                                   << "[" << (p4[0])/512/S[1] << ", " 
                                   <<  (p4[1])/512/S[2] << ", "
                                   <<  (p4[2])/426/S[3] << "]" << std::endl;   */     
	//stim::vec<float> tempPos = spidey.getPosition();
	//stim::vec<float> tempDir = spidey.getDirection();
	//stim::vec<float> tempMag = spidey.getMagnitude();
	//spidey.setPosition(tempPos[0]+-tempDir[0]*tempMag[0]/2,
	//			tempPos[1]+-tempDir[1]*tempMag[0]/2,
	//			tempPos[2]+-tempDir[2]*tempMag[0]/2);
	     //spidey.findOptimalDirection();
	//spidey.Update();

	   
}


void 
changeSize(int w, int h)
{
        glViewport(0,0,w,h);
        glMatrixMode(GL_PROJECTION);
        glLoadIdentity();
	glOrtho(-5.0, 2.0,-5.0, 2.0, -0.0, 1000.0);
	//gluPerspective(90, 4.0/3.0, 0.1, 100.0);
        glMatrixMode(GL_MODELVIEW);
}


int
main(int argc, char **argv)
{
	CHECK_OPENGL_ERROR
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(size[0], size[1]);
	glutCreateWindow("gl_texture");

	glutKeyboardFunc(processKeys);
	glutSpecialFunc(processSpecialKeys);
        glutReshapeFunc(changeSize);
        glutDisplayFunc(renderScene);
	glutMouseFunc(MouseButton);
	glutMotionFunc(MouseMotion);

	GLenum err = glewInit();
	if (GLEW_OK != err)
		{
			std::cerr << "Failed" << std::endl;
			fprintf(stderr, "Error: %s\n", glewGetErrorString(err));
		}
//	stim::gl_texture<unsigned char> stack ("/media/pavel/Storage/Data/Test_out/4_3/");
	stim::gl_texture<unsigned char> stack ("/home/pavel/Documents/Test_Data/");
//	stim::gl_texture<unsigned char> stack ("/media/pavel/Storage/Data/Partitioned/2008_ii_Clean_parts/4_4/a0-1023");
	stack.setTexParam(GL_LINEAR, GL_CLAMP_TO_EDGE, GL_UNSIGNED_BYTE, GL_LUMINANCE);
	//stack.setTexParam(GL_LINEAR, GL_CLAMP_TO_EDGE, GL_FLOAT, GL_LUMINANCE);
//	stack.setDimensions(VOXDIMS[0],VOXDIMS[1],VOXDIMS[2]);
	//D = stack.getDimensions();
	stack.createTexture();
	texID = stack.getTexture();
	start = std::clock();


	glInit();
	glutIdleFunc(idleFunction);
	glutMainLoop();
	return 0;
}


